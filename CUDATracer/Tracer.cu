#include "hip/hip_runtime.h"
#include "Tracer.cuh"

#define EPS 1e-3
#define BACKGROUND make_float4(0.0f, 0.0f, 0.0f, 1.0f)

#define BASE_SPHERE make_float3(0.0f, 0.0f, -8.0f)
#define BASE_SPHERE_RADIUS 2.0f

__constant__ float4 mWorldPosition[1];
__constant__ float4 mWorldMatrix[4];

__device__ float VectorDot(float4 A, float4 B) 
{
	return A.x * B.x +
		A.y * B.y +
		A.z * B.z +
		A.w * B.w;
}

__device__ float VectorDot(float3 A, float3 B)
{
	return A.x * B.x +
		A.y * B.y +
		A.z * B.z;
}

__device__ float4 VectorSub(float4 A, float4 B) 
{
	return make_float4(A.x - B.x,
		A.y - B.y,
		A.z - B.z,
		A.w - B.w);
}

__device__ float3 VectorSub(float3 A, float3 B)
{
	return make_float3(
		A.x - B.x,
		A.y - B.y,
		A.z - B.z
		);
}

__device__ float3 NormalizeVector(float3 A) 
{
	float d = A.x * A.x + A.y * A.y + A.z * A.z;
	d = sqrt(d);
	return make_float3(
		A.x / d,
		A.y / d,
		A.z / d
	);
}

__device__ float4 BuildDirection(float4 mSource) 
{
	float4 mDestDirection = make_float4(
		VectorDot(mWorldMatrix[0], mSource),
		VectorDot(mWorldMatrix[1], mSource),
		VectorDot(mWorldMatrix[2], mSource),
		VectorDot(mWorldMatrix[3], mSource));

	mDestDirection.x /= mDestDirection.w;
	mDestDirection.y /= mDestDirection.w;
	mDestDirection.z /= mDestDirection.w;
	mDestDirection.w = 0.0f;

	return mDestDirection;
}

__device__ float SolveQU(float a, float b, float c) 
{
	float tx = -c / b;
	if (a == 0.0f)
		return tx >= 1.0f ? tx : 0.0f;

	float D = b * b - 4.0f * a * c;

	if (D < 0.0f)
		return 0.0f;

	float DSQ = sqrt(D);
	float x1 = (-b + DSQ) / (2.0f * a);
	float x2 = (-b - DSQ) / (2.0f * a);
	
	x1 = x1 >= 1.0f ? x1 : 0.0f;
	x2 = x2 >= 1.0f ? x2 : 0.0f;
	return min(x1, x2);
}

__device__ float3 MakePointFromLine(float4 mRayStart, float4 mRayDir, float mT) 
{
	return make_float3(
		mRayStart.x + mRayDir.x * mT,
		mRayStart.y + mRayDir.y * mT,
		mRayStart.z + mRayDir.z * mT
	);
}

__device__ float4 IntersectRaySphere(float4 mRayStart, float4 mRayDir)
{

	float3 mSphere = BASE_SPHERE;
	float3 mSub = make_float3(
		mRayStart.x - mSphere.x,
		mRayStart.y - mSphere.y,
		mRayStart.z - mSphere.z
	);

	float a, b, c;
	a = VectorDot(mRayDir, mRayDir);//mRayDir.x * mRayDir.x + mRayDir.y *mRayDir.y + mRayDir.z * mRayDir.z;
	b = 2.0f * VectorDot(
		make_float3(mRayDir.x, mRayDir.y, mRayDir.z),
		mSub);//mRayDir.x * mSub.x + 
	c = VectorDot(mSub, mSub) - BASE_SPHERE_RADIUS * BASE_SPHERE_RADIUS;

	float mT = SolveQU(a, b, c);
	if(mT < 1.0f)
		return BACKGROUND;

	float3 mSphereIntrsectPoint = MakePointFromLine(mRayStart, mRayDir, mT);
	float3 mSpherePointNormal = VectorSub(mSphereIntrsectPoint, BASE_SPHERE);
	mSpherePointNormal = NormalizeVector(mSpherePointNormal);

	//mSpherePointNormal.x = mSpherePointNormal.x * 0.5f + 0.5f;
	//mSpherePointNormal.y = mSpherePointNormal.y * 0.5f + 0.5f;
	//mSpherePointNormal.z = mSpherePointNormal.z * 0.5f + 0.5f;

	return make_float4(
		mSpherePointNormal.x,
		mSpherePointNormal.y,
		mSpherePointNormal.z,
		1.0f);
}

__global__ void RayTrace(float4 *mTextureBuffer, int32_t mDimensionX, int32_t mDimensionY)
{
	int gDispatchX = blockIdx.x * blockDim.x + threadIdx.x;
	int mDispatchY = blockIdx.y * blockDim.y + threadIdx.y;

	if (gDispatchX < mDimensionX && mDispatchY < mDimensionY)
	{
		float4 mRay = make_float4((float)gDispatchX / mDimensionX, (float)mDispatchY / mDimensionY, -1.0f, 1.0f);
		mRay.x = 2.0f * mRay.x - 1.0f;
		mRay.y = 2.0f * mRay.y - 1.0f;

		float4 mRayDir = BuildDirection(mRay);
		float4 mRayStart = mWorldPosition[0];
		mRayDir = VectorSub(mRayDir, mRayStart);

		//float ln = sqrt(mRayDir.x * mRayDir.x + mRayDir.y * mRayDir.y + mRayDir.z * mRayDir.z);

		//float mValue = pow(VectorDot(mRayDir, make_float4(0.0f, 0.0f, -1.0f, 0.0f)) / ln, 8.0f);

		mTextureBuffer[mDispatchY * mDimensionX + gDispatchX] = IntersectRaySphere(mRayStart, mRayDir);
			//make_float4(mValue, mValue, mValue, 1.0f);
			//(float)gDispatchX / mDimensionX, (float)mDispatchY / mDimensionY, 1.0f, 1.0f);
	}
}


TracerCUDA::Tracer::Tracer(const TraceInfo &mInfo) : mInfo(mInfo)
{
	hipDeviceProp_t cudaDevice_prop;
	memset(&cudaDevice_prop, 0, sizeof(cudaDevice_prop));
	cudaDevice_prop.major = 1;
	cudaDevice_prop.minor = 3;

	CUDA_HR(hipChooseDevice(&cudaGLDevice_ID, &cudaDevice_prop));
	CUDA_HR(cudaGLSetGLDevice(cudaGLDevice_ID));
}

TracerCUDA::Tracer::~Tracer()
{
	CUDA_HR(hipGraphicsUnregisterResource(mCUDA_texture));
}

void TracerCUDA::Tracer::Resize(const TraceInfo &mInfo)
{
	this->mInfo = mInfo;
}

void TracerCUDA::Tracer::InitTexture(void)
{
	CUDA_HR(hipGraphicsGLRegisterBuffer(&mCUDA_texture, mInfo.mTextureBuffer, cudaGraphicsMapFlagsNone));
}

void TracerCUDA::Tracer::MapResource(void)
{
	CUDA_HR(hipGraphicsMapResources(1, &mCUDA_texture, 0));

	size_t mSize = mInfo.mSizeTextureBuffer;
	CUDA_HR(hipGraphicsResourceGetMappedPointer((void **)&mCUDA_BUFFER, &mSize, mCUDA_texture));
}

void TracerCUDA::Tracer::UnmapResource(void)
{
	CUDA_HR(hipDeviceSynchronize());

	CUDA_HR(hipGraphicsUnmapResources(1, &mCUDA_texture, 0));
}

void TracerCUDA::Tracer::Trace(float *mPosition, float *mMatrix)
{
	float mPos[4] = { mPosition[0], mPosition[1], mPosition[2], 0.0f };

	hipMemcpyToSymbol(HIP_SYMBOL(mWorldPosition), mPos, sizeof(float4));
	hipMemcpyToSymbol(HIP_SYMBOL(mWorldMatrix), mMatrix, 4 * sizeof(float4));

	dim3 threads(32, 32);
	dim3 blocks((mInfo.mDimX + 31) / 32, (mInfo.mDimY + 31) / 32);
	RayTrace <<<blocks, threads >>> (mCUDA_BUFFER, mInfo.mDimX, mInfo.mDimY);
}